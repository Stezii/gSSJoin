#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2016 by Sidney Ribeiro Junior
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <vector>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include <map>

#include "structs.cuh"
#include "utils.cuh"
#include "inverted_index.cuh"
#include "simjoin.cuh"


#define OUTPUT 1
#define NUM_STREAMS 1


using namespace std;

struct FileStats {
	int num_sets;
	int num_terms;

	vector<int> sizes; // set sizes
	vector<float> weighted_sizes; // weighted set sizes
	vector<int> start; // beginning of each entry
	vector<float> token_weights; // weights of each token

	FileStats() : num_sets(0), num_terms(0) {}
};

FileStats readInputFiles(string &sets_filename, string &weights_filename, vector<Entry> &entries, vector<string> &ids);
void processTestFile(InvertedIndex &index, FileStats &stats, string &file, vector<string> &ids, float threshold, int topk, bool topk_is_strict, stringstream &fileout);


/**
 * Receives as parameters the training file name and the test file name
 */

static int num_tests = 0;
int biggestQuerySize = -1;


int main(int argc, char **argv) {

	if (argc != 8) {
		cerr << "Wrong parameters. Correct usage: <executable> <input_token_file> <input_weights_file> <threshold> <topk> <topk_mode> <output_file> <number_of_gpus>" << endl;
		exit(1);
	}

	bool topk_is_strict;
	string topk_mode(argv[5]);
	if (topk_mode == "strict") {
		topk_is_strict = true;
	} else if (topk_mode == "soft") {
		topk_is_strict = false;
	} else {
		cerr << "Wrong parameter 'topk_mode'. Must be 'strict' or 'soft'" << endl;
		exit(1);
	}

	int gpuNum;
	hipGetDeviceCount(&gpuNum);

	if (gpuNum > atoi(argv[7])) {
		gpuNum = atoi(argv[7]);
		if (gpuNum < 1)
			gpuNum = 1;
	}
	//cerr << "Using " << gpuNum << "GPUs" << endl;

	// we use 2 streams per GPU
	int numThreads = gpuNum*NUM_STREAMS;

	omp_set_num_threads(numThreads);

#if OUTPUT
	//truncate output files
	ofstream ofsf(argv[6], ofstream::trunc);
	ofsf.close();

	ofstream ofsfileoutput(argv[6], ofstream::out | ofstream::app);
#endif
	vector<string> inputs;// to read the whole test file in memory
	vector<InvertedIndex> indexes;
	indexes.resize(gpuNum);

	double starts, ends;

	string inputSetsFileName(argv[1]);
	string inputWeightsFileName(argv[2]);

	printf("Reading files...\n");
	vector<Entry> entries;
	vector<string> ids;

	starts = gettime();
	FileStats stats = readInputFiles(inputSetsFileName, inputWeightsFileName, entries, ids);
	ends = gettime();

	printf("Time taken: %lf seconds\n", ends - starts);

	vector<stringstream*> outputString;
	//Each thread builds an output string, so it can be flushed at once at the end of the program
	for (int i = 0; i < numThreads; i++) {
		outputString.push_back(new stringstream);
	}

	//create an inverted index for all streams in each GPU
	#pragma omp parallel num_threads(gpuNum)
	{
		int cpuid = omp_get_thread_num();
		hipSetDevice(cpuid);
		double start, end;

		start = gettime();
		indexes[cpuid] = make_inverted_index(stats.num_sets, stats.num_terms, entries);
		end = gettime();

		#pragma omp single nowait
		printf("Total time taken for insertion: %lf seconds\n", end - start);
	}


	#pragma omp parallel
	{
		int cpuid = omp_get_thread_num();
		hipSetDevice(cpuid / NUM_STREAMS);

		float threshold = atof(argv[3]);
		float topk = atof(argv[4]);

		FileStats lstats = stats;

		processTestFile(indexes[cpuid / NUM_STREAMS], lstats, inputSetsFileName, ids, threshold, topk, topk_is_strict, *outputString[cpuid]);
		if (cpuid %  NUM_STREAMS == 0)
			gpuAssert(hipDeviceReset());

	}

#if OUTPUT
		starts = gettime();
		for (int i = 0; i < numThreads; i++) {
			ofsfileoutput << outputString[i]->str();
		}
		ends = gettime();

		printf("Time taken to write output: %lf seconds\n", ends - starts);

		ofsfileoutput.close();
#endif
		return 0;
}

FileStats readInputFiles(string &sets_filename, string &weights_filename, vector<Entry> &entries, vector<string> &ids) {
	string line;
	FileStats stats;

	// get number of terms and check weights file
	ifstream input_weights(weights_filename.c_str());
	stats.num_terms = 1; // must start at 1 for zero-based array access

	while (!input_weights.eof()) {
		getline(input_weights, line);
		if (line == "") continue;

		vector<string> line_spl = split(line, ' ');
		int token = atoi(line_spl[0].c_str());
		float weight = atof(line_spl[1].c_str());
		if (stats.num_terms == 1 && token != 1) {
			cerr << "Error in " << weights_filename << ": First token id must be 1 in " << endl;
			exit(1);
		}
		if (stats.num_terms != token) {
			cerr << "Error in " << weights_filename << ": Token " << stats.num_terms << " is missing" << endl;
			exit(1);
		}
		if (weight < 0) {
			cerr << "Error in " << weights_filename << ": Token weight may not be smaller than 0" << endl;
			exit(1);
		}
		stats.num_terms++;
	}

	// read weights
	input_weights.clear();
	input_weights.seekg(0, ios::beg);
	float token_weights[stats.num_terms];

	while (!input_weights.eof()) {
		getline(input_weights, line);
		if (line == "") continue;

		vector<string> line_spl = split(line, ' ');
		int token = atoi(line_spl[0].c_str());
		float weight = atof(line_spl[1].c_str());
		token_weights[token] = weight;
	}

	input_weights.close();

	vector<float> vec(token_weights, token_weights + stats.num_terms);
	stats.token_weights = vec;

	// read sets
	ifstream input_sets(sets_filename.c_str());
	int accumulatedsize = 0;
	int set_id = 0;

	while (!input_sets.eof()) {
		getline(input_sets, line);
		if (line == "") continue;

		vector<string> line_spl = split(line, ' ');
		vector<string> tokens(line_spl.begin() + 1, line_spl.begin() + (int)line_spl.size());
		ids.push_back(line_spl[0]);

		biggestQuerySize = max((int)tokens.size(), biggestQuerySize);

		int size = tokens.size();
		float weighted_size = 0;
		stats.sizes.push_back(size);
		stats.start.push_back(accumulatedsize);
		accumulatedsize += size;

		for (int i = 0; i < size; i++) {
			int term_id = atoi(tokens[i].c_str());
			entries.push_back(Entry(set_id, term_id));
			weighted_size += stats.token_weights.at(term_id);
		}
		stats.weighted_sizes.push_back(weighted_size);
		set_id++;
	}

	stats.num_sets = stats.start.size();

	input_sets.close();

	return stats;
}

void allocVariables(DeviceVariables *dev_vars, float threshold, int num_sets, int num_terms, Similarity** distances) {
	dim3 grid, threads;

	get_grid_config(grid, threads);

	gpuAssert(hipMalloc(&dev_vars->d_dist, num_sets * sizeof(Similarity))); // distance between all the sets and the query doc
	gpuAssert(hipMalloc(&dev_vars->d_result, num_sets * sizeof(Similarity))); // compacted similarities between all the sets and the query doc
	gpuAssert(hipMalloc(&dev_vars->d_sim, num_sets * sizeof(float))); // count of elements in common
	gpuAssert(hipMalloc(&dev_vars->d_wsizes, num_sets * sizeof(float))); // weighted size of all sets
	gpuAssert(hipMalloc(&dev_vars->d_tokweights, num_terms * sizeof(float))); // weights of each token
	gpuAssert(hipMalloc(&dev_vars->d_query, biggestQuerySize * sizeof(Entry))); // query
	gpuAssert(hipMalloc(&dev_vars->d_index, biggestQuerySize * sizeof(int)));
	gpuAssert(hipMalloc(&dev_vars->d_count, biggestQuerySize * sizeof(int)));

	*distances = (Similarity*)malloc(num_sets * sizeof(Similarity));

	int blocksize = 1024;
	int numBlocks = num_sets / blocksize + (num_sets % blocksize ? 1 : 0);

	gpuAssert(hipMalloc(&dev_vars->d_bC,sizeof(int)*(numBlocks + 1)));
	gpuAssert(hipMalloc(&dev_vars->d_bO,sizeof(int)*numBlocks));

}

void freeVariables(DeviceVariables *dev_vars, InvertedIndex &index, Similarity** distances) {
	hipFree(dev_vars->d_dist);
	hipFree(dev_vars->d_result);
	hipFree(dev_vars->d_sim);
	hipFree(dev_vars->d_wsizes);
	hipFree(dev_vars->d_tokweights);
	hipFree(dev_vars->d_query);
	hipFree(dev_vars->d_index);
	hipFree(dev_vars->d_count);
	hipFree(dev_vars->d_bC);
	hipFree(dev_vars->d_bO);

	free(*distances);

	if (omp_get_thread_num() % NUM_STREAMS == 0) {
		hipFree(index.d_count);
		hipFree(index.d_index);
		hipFree(index.d_inverted_index);
	}
}

void processTestFile(InvertedIndex &index, FileStats &stats, string &filename, vector<string> &ids, float threshold, int topk, bool topk_is_strict, stringstream &outputfile) {

	int num_test_local = 0, setid;

	//#pragma omp single nowait
	printf("Processing input file %s...\n", filename.c_str());

	DeviceVariables dev_vars;
	Similarity* distances;

	allocVariables(&dev_vars, threshold, index.num_sets, stats.num_terms, &distances);

	hipMemcpyAsync(dev_vars.d_wsizes, &stats.weighted_sizes[0], index.num_sets * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_vars.d_tokweights, &stats.token_weights[0], stats.num_terms * sizeof(float), hipMemcpyHostToDevice);

	double start = gettime();

#pragma omp for
	for (setid = 0; setid < index.num_sets - 1; setid++) {

		num_test_local++;

		int totalSimilars = findSimilars(index, threshold, topk, topk_is_strict, &dev_vars, distances, setid, stats.start[setid], stats.sizes[setid], stats.weighted_sizes[setid]);

#if OUTPUT
		for (int i = 0; i < totalSimilars; i++) {
			outputfile << ids[setid] << "\t" << ids[distances[i].set_id] << "\t" << distances[i].similarity << endl;
		}
#endif

	}

	freeVariables(&dev_vars, index, &distances);
	int threadid = omp_get_thread_num();

	printf("Entries in device %d stream %d: %d\n", threadid / NUM_STREAMS, threadid %  NUM_STREAMS, num_test_local);

	#pragma omp barrier

	double end = gettime();

	#pragma omp master
	{
		printf("Time taken for %d queries: %lf seconds\n\n", num_tests, end - start);
	}
}
