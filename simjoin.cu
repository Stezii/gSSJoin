#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2015 by Wisllay Vitrio
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

/* *
 * knn.cu
 */

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <queue>
#include <vector>
#include <set>
#include <functional>

#include "simjoin.cuh"
#include "structs.cuh"
#include "utils.cuh"
#include "inverted_index.cuh"
#include "cuCompactor.cuh"


struct is_bigger_than_threshold
{
	float threshold;
	is_bigger_than_threshold(float thr) : threshold(thr) {};
	__host__ __device__
	bool operator()(const Similarity &reg)
	{
		return (reg.similarity > threshold);
	}
};

__host__ int findSimilars(InvertedIndex inverted_index, float threshold, struct DeviceVariables *dev_vars, Similarity* h_result,
		int querybegin, int queryqtt) {

	dim3 grid, threads;
	get_grid_config(grid, threads);

	int num_docs = inverted_index.num_docs;
	int *intersection = dev_vars->d_intersection, *sizes = dev_vars->d_sizes, *starts = dev_vars->d_starts;
	Entry *query = inverted_index.d_entries;
	Similarity *d_result = dev_vars->d_result;
	Similarity *d_similarity = dev_vars->d_similarity;
	int totalSimilars = 0;

	int *d_BlocksCount = dev_vars->d_bC, *d_BlocksOffset = dev_vars->d_bO;

	gpuAssert(hipMemset(intersection, 0,(1 + queryqtt*num_docs)*sizeof(int)));

	calculateIntersection<<<grid, threads>>>(inverted_index, query, intersection, querybegin, queryqtt, starts, sizes, threshold);

	calculateSimilarity<<<grid, threads>>>(d_similarity, intersection, sizes, querybegin, num_docs, queryqtt);

	int blocksize = 1024;
	int numBlocks = cuCompactor::divup(num_docs*queryqtt, blocksize);
	totalSimilars = cuCompactor::compact2<Similarity>(d_result, d_similarity, num_docs*queryqtt, is_bigger_than_threshold(threshold), blocksize, numBlocks, d_BlocksCount, d_BlocksOffset);

	return totalSimilars;
}

__global__ void calculateIntersection(InvertedIndex inverted_index, Entry *query, int *intersection, int querybegin, int queryqtt,
		int *docstart, int *docsizes, float threshold) {

	int block_start, block_end, docid, size, maxsize;

	for (int q = 0; q < queryqtt && q < inverted_index.num_docs - 1; q++) { // percorre as queries

		docid = querybegin + q;
		size = docsizes[docid];
		maxsize = ceil(((float) size)/threshold) + 1;

	for (int idx = blockIdx.x; idx < size; idx += gridDim.x) { // percorre os termos da query (apenas os que estão no midprefix)
			Entry entry = query[idx + docstart[docid]]; // find the term

			block_start = entry.term_id == 0 ? 0 : inverted_index.d_index[entry.term_id-1];
			block_end = inverted_index.d_index[entry.term_id];

			for (int i = block_start + threadIdx.x; i < block_end; i += blockDim.x) { // percorre os documentos que tem aquele termo
				Entry index_entry = inverted_index.d_inverted_index[i]; // obter item

				// somar na distância
				if (index_entry.doc_id > docid && docsizes[index_entry.doc_id] < maxsize) {
					atomicAdd(&intersection[q*inverted_index.num_docs + index_entry.doc_id], 1);
				}
			}
		}
	}
}

__global__ void calculateSimilarity(Similarity *similarity, int *intersection, int *sizes, int begin, int num_docs, int queryqtt) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (; i < num_docs*queryqtt; i += blockDim.x * gridDim.x) {
		similarity[i].doc_i = begin + i/num_docs;
		similarity[i].doc_j = i%num_docs;
		similarity[i].similarity = ((float) intersection[i])/((float) sizes[similarity[i].doc_i] + sizes[similarity[i].doc_j] - intersection[i]);
	}
}
